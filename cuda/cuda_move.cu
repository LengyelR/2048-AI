#include "hip/hip_runtime.h"
#include "cuda_move.cuh"  // TODO: hip/hip_runtime.h, hip/hip_runtime.h, , etc...
#include "bit_move.h"

#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include "hip/hip_cooperative_groups.h"

#define GET_CELL(board, idx) (board & cell_masks[idx]) >> (15 - idx) * 4

namespace cg = cooperative_groups;

namespace cuda_sim 
{

    int search_move(const uint64_t board)
    {
        //hard coded params, cython doesn't know template args
        const int stop = 300;
        const int grid_size = 2048;
        const int block_size = 512;
        const int N = grid_size * block_size;

        auto boards = std::make_unique<uint64_t[]>(N);

        uint64_t current_score = 0;
        uint64_t best_score = 0;
        int best_move = 0;

        //note: streaming doesn't help because this already achieves near 100% occupancy...
        for (int move = 0; move < 4; ++move)
        {
            auto moved_board = simulator::do_move(board, move);
            std::fill_n(boards.get(), N, moved_board);

            random_move<grid_size, block_size, stop>(boards.get());
            current_score = get_score<grid_size, block_size>(boards.get());

            if (best_score < current_score)
            {
                best_score = current_score;
                best_move = move;
            }
        }

        return best_move;
    }

    int search_move(const std::vector<std::vector<int>>& vec_board)
    {
        auto board = convert::to_bits(vec_board[0], vec_board[1], vec_board[2], vec_board[3]);
        return search_move(board);
    }

    __device__ const uint64_t cell_masks[16]
    {
        0xf000000000000000,
        0x0f00000000000000,
        0x00f0000000000000,
        0x000f000000000000,
        0x0000f00000000000,
        0x00000f0000000000,
        0x000000f000000000,
        0x0000000f00000000,
        0x00000000f0000000,
        0x000000000f000000,
        0x0000000000f00000,
        0x00000000000f0000,
        0x000000000000f000,
        0x0000000000000f00,
        0x00000000000000f0,
        0x000000000000000f
    };

    namespace utils
    {
        // could have used intrinsics for these 2
        // TODO: does it work properly for 16 bits?
        static __device__ __inline__ uint32_t __popcnt(const uint32_t x)
        {
            uint32_t count;
            asm volatile("popc.b32 %0, %1;" : "=r"(count) : "r"(x));
            return count;
        }

        static __device__ __inline__ uint32_t __myclock()
        {
            uint32_t mclk;
            asm volatile("mov.u32 %0, %%clock;" : "=r"(mclk));
            return mclk;
        }

        /// draws random number in range [1, upper_bound] (both inclusive)
        __device__ int get_random_int(hiprandState* state, int upper_bound)
        {
            hiprand_init(__myclock(), 0, 0, state);
            auto rand = hiprand_uniform(state);
            return static_cast<int>(rand*upper_bound + 0.999999);
        }
    }

    namespace cumove
    {
        __device__ int get_tile_position(uint16_t zero_cells, int rand_idx)
        {
            int counter = 0;
            for (int i = 0; i < 16; ++i)
            {
                if (zero_cells & (1 << i))
                {
                    counter++;
                }

                if (counter == rand_idx)
                {
                    return i;
                }
            }

            return -1;
        }

        __device__ uint64_t place_random(uint64_t board)
        {
            //todo: any faster?
            //uint16_t zero_cells = 0;
            //#pragma unroll  
            //for (int i = 0; i < 16; ++i)
            //{
            //    zero_cells |= (1 << i) * !!(board & cell_masks[i]);
            //}
            //zero_cells = ~zero_cells;

            uint16_t zero_cells = 0;
            for (int i = 0; i < 16; ++i)
            {
                if (board & cell_masks[i])
                {
                    continue;
                }
                else
                {
                    zero_cells |= 1 << i;
                }
            }

            if (zero_cells == 0)
            {
                return board;
            }

            int range_end = utils::__popcnt(zero_cells);

            hiprandState state;
            auto rand_idx = utils::get_random_int(&state, range_end);

            auto idx = get_tile_position(zero_cells, rand_idx);
            auto new_cell = hiprand_uniform(&state) > 0.9 ? 2ULL : 1ULL;

            return board | (new_cell << (60 - 4 * idx));
        }

        __device__ static uint16_t row_right[0xffff];
        __device__ static uint16_t row_left[0xffff];

        const uint64_t ROW_MASK0 = 0xffff000000000000;
        const uint64_t ROW_MASK1 = 0x0000ffff00000000;
        const uint64_t ROW_MASK2 = 0x00000000ffff0000;
        const uint64_t ROW_MASK3 = 0x000000000000ffff;

        __device__ uint64_t do_move(uint64_t board, int move)
        {
            switch (move)
            {
            case 0:
                board = board_up(board);
                break;
            case 1:
                board = board_down(board);
                break;
            case 2:
                board = board_left(board);
                break;
            case 3:
                board = board_right(board);
                break;
            }

            return board;
        }

        //todo: make it __host__ __device__ 
        __device__ uint64_t transpose(uint64_t x)
        {
            uint64_t a1 = x & 0xf0f00f0ff0f00f0fULL;
            uint64_t a2 = x & 0x0000f0f00000f0f0ULL;
            uint64_t a3 = x & 0x0f0f00000f0f0000ULL;
            uint64_t a = a1 | (a2 << 12) | (a3 >> 12);
            uint64_t b1 = a & 0xff00ff0000ff00ffULL;
            uint64_t b2 = a & 0x00ff00ff00000000ULL;
            uint64_t b3 = a & 0x00000000ff00ff00ULL;
            return b1 | (b2 >> 24) | (b3 << 24);
        }


        template<uint16_t row_direction[0xffff]>
        __device__ uint64_t board_move(uint64_t board)
        {
            uint16_t row0 = (board & ROW_MASK0) >> 48;
            uint16_t row1 = (board & ROW_MASK1) >> 32;
            uint16_t row2 = (board & ROW_MASK2) >> 16;
            uint16_t row3 = (board & ROW_MASK3);

            auto res0 = static_cast<uint64_t>(row_direction[row0]) << 48;
            auto res1 = static_cast<uint64_t>(row_direction[row1]) << 32;
            auto res2 = static_cast<uint64_t>(row_direction[row2]) << 16;
            auto res3 = static_cast<uint64_t>(row_direction[row3]);

            return res0 | res1 | res2 | res3;
        }

        __device__ uint64_t board_right(uint64_t board)
        {
            return board_move<row_right>(board);
        }

        __device__ uint64_t board_left(uint64_t board)
        {
            return board_move<row_left>(board);
        }

        __device__ uint64_t board_up(uint64_t board)
        {
            auto transposed = transpose(board);
            auto moved = board_left(transposed);
            return transpose(moved);
        }

        __device__ uint64_t board_down(uint64_t board)
        {
            auto transposed = transpose(board);
            auto moved = board_right(transposed);
            return transpose(moved);
        }

    }

    __device__ uint64_t score(uint64_t board)
    {
        uint64_t score = 0;
        for (int i = 0; i < 16; ++i)
        {
            auto cell = GET_CELL(board, i);
            score += ((2 << (cell - 1)) & ~1ULL);
        }
        return score;
    }

    void init()
    {
        move::init();
        hipMemcpyToSymbol(HIP_SYMBOL(cumove::row_right), move::row_right, 0xffff * sizeof(uint16_t));
        hipMemcpyToSymbol(HIP_SYMBOL(cumove::row_left), move::row_left, 0xffff * sizeof(uint16_t));
    }

    __global__ void move_kernel(const uint64_t* boards, int* moves, uint64_t* moved_boards)
    {
        int tidx = blockIdx.x * blockDim.x + threadIdx.x;
        auto moved_board = cumove::do_move(boards[tidx], moves[tidx]);
        moved_boards[tidx] = moved_board == boards[tidx] 
                                ? moved_board 
                                : cumove::place_random(moved_board);
    }

    __device__ uint64_t new_board()
    {
        auto new_board = cumove::place_random(0);
        return cumove::place_random(new_board);
    }
    __global__ void new_board_kernel(uint64_t* boards)
    {
        int tidx = blockIdx.x * blockDim.x + threadIdx.x;
        boards[tidx] = new_board();
    }
    
    __global__ void random_move_kernel(uint64_t* boards, const int stop)
    {
        hiprandState state;
        int tidx = blockIdx.x * blockDim.x + threadIdx.x;

        for (int i = 0; i < stop; i++)
        {
            auto copy = boards[tidx];

            auto rand_move = (utils::get_random_int(&state, 4)-1);
            auto moved_board = cumove::do_move(boards[tidx], rand_move);
            boards[tidx] = moved_board == copy
                ? moved_board
                : cumove::place_random(moved_board);
        }
    }

    __global__ void reduce_score_kernel(uint64_t* boards, uint64_t* output)
    {
        cg::thread_block cta = cg::this_thread_block();
        extern __shared__ uint64_t sdata[];

        unsigned int tid = threadIdx.x;
        unsigned int i = blockIdx.x*blockDim.x * 2 + threadIdx.x;
        unsigned int gridSize = blockDim.x * 2 * gridDim.x;
        unsigned int n = blockDim.x * gridDim.x;
        uint64_t mySum = 0;

        while (i < n)
        {
            mySum += score(boards[i]);
            mySum += score(boards[i + blockDim.x]);
            i += gridSize;
        }

        sdata[tid] = mySum;
        cg::sync(cta);

        if (tid < 256)
        {
            sdata[tid] = mySum = mySum + sdata[tid + 256];
        }
        cg::sync(cta);

        if (tid < 128)
        {
            sdata[tid] = mySum = mySum + sdata[tid + 128];
        }
        cg::sync(cta);

        if (tid <  64)
        {
            sdata[tid] = mySum = mySum + sdata[tid + 64];
        }
        cg::sync(cta);

        auto tile32 = cg::tiled_partition<32>(cta);
        if (cta.thread_rank() < 32)
        {
            mySum += sdata[tid + 32];
            for (int offset = tile32.size() / 2; offset > 0; offset /= 2)
            {
                mySum += tile32.shfl_down(mySum, offset);
            }
        }

        if (cta.thread_rank() == 0)
        {
            output[blockIdx.x] = mySum;
        }
    }
}